#include "hip/hip_runtime.h"
#include "RasterParallelAlgorithm.h"
#include "RasterUnitFunction.cuh"
#include "RasterParallelAlgorithm.cuh"

namespace cg = cooperative_groups;

namespace
{
	unsigned int* dBlockSumBuffer = nullptr; // warp sum buffer used in inter-block scan, set to 1024 as default
	unsigned int* dSum = nullptr;

	constexpr int blockSize = 256;
}



// suggest block size 256 -> can compact ~ 500 thousands primitives
__global__ void PrimitiveCompactionKernel(int size, const Primitive* inputStream, Primitive* outputStream, unsigned int* gBlockScanBuffer , unsigned int* sum)
{
	__shared__ unsigned int sWarpScanBuffer[32];
	const unsigned int leaderLane = 0;

	cg::grid_group grid = cg::this_grid();

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size && blockIdx.x != 0) return;

	bool predicate = idx < size ? (inputStream[idx].primitiveID != -1) : false;
	unsigned int ballotMask = __ballot_sync(0xFFFFFFFF, predicate);
	
	unsigned int  lane_id = get_lane_id();
	unsigned int lowerBits = ballotMask & ((1U << lane_id) - 1);
	unsigned int offsetInWarp = __popc(ballotMask & lowerBits );

	unsigned int totalValidInWarp = __popc(ballotMask);

	if (threadIdx.x < 32)
	{
		sWarpScanBuffer[threadIdx.x] = 0;
	}
	__syncthreads();

	if (lane_id == leaderLane)
	{
		unsigned int warpIndex = threadIdx.x / 32;
		sWarpScanBuffer[warpIndex] = totalValidInWarp;
	}
	__syncthreads();

	if ( threadIdx.x < 32 ) //first warp
	{	
		unsigned int warpSum = sWarpScanBuffer[threadIdx.x];
		unsigned int warpPrefixSum = ScanInWarpExclusive(warpSum);
		sWarpScanBuffer[threadIdx.x] = warpPrefixSum;
		if (threadIdx.x == (blockDim.x + 31) / 32) gBlockScanBuffer[blockIdx.x] = warpPrefixSum + warpSum;
	}
	__syncthreads();
	unsigned int offsetInBlock = sWarpScanBuffer[threadIdx.x / 32] + offsetInWarp;

	grid.sync();

	if(blockIdx.x == 0) // the second pass only use the first block with every thread read uint4 vector to reduce
	{
		if (threadIdx.x < 32) sWarpScanBuffer[threadIdx.x] = 0;
		__syncthreads();
		uint4 idata = ((uint4*)gBlockScanBuffer)[threadIdx.x];
		uint4 odata = ScanInBlock4Exclusive(idata, sWarpScanBuffer);
		((uint4*)gBlockScanBuffer)[threadIdx.x] = odata;
		__syncthreads();
		if (threadIdx.x == 0)
		{
			*sum = gBlockScanBuffer[1023];
		}
	}
	__syncthreads();

	grid.sync();

	if (predicate)
	{
		unsigned int blockPrefix = gBlockScanBuffer[blockIdx.x];
		unsigned int outputIndex = blockPrefix + offsetInBlock;
		outputStream[outputIndex] = inputStream[idx];
	}
}

void InitCompactionEnvironment()
{
	hipMalloc((void**)&dBlockSumBuffer, sizeof(unsigned int) * 1024);
	CUDA_CHECK(hipMalloc((void**)&dSum, sizeof(unsigned int)));
}

void  PrimitiveCompaction(int size, const Primitive* inputStream, Primitive* outputStream, unsigned int* sum, hipStream_t stream)
{
	int numBlocks = (size + blockSize - 1) / blockSize;
	CUDA_CHECK(hipMemsetAsync(dBlockSumBuffer, 0, sizeof(unsigned int) * 1024, stream));
	CUDA_CHECK(hipMemsetAsync(dSum, 0, sizeof(unsigned int), stream));
	
	void* args[] = { &size, &inputStream, &outputStream, &dBlockSumBuffer, &dSum };
	hipLaunchCooperativeKernel(reinterpret_cast<const void*>(PrimitiveCompactionKernel),numBlocks, blockSize, args, std::max(blockSize / 32u, 32u) * sizeof(unsigned int),stream);
	CUDA_CHECK(hipMemcpyAsync(sum, dSum, sizeof(unsigned int), hipMemcpyDeviceToDevice, stream));
}

void DestroyCompactionEnvironment()
{
	CUDA_CHECK(hipFree(dBlockSumBuffer));
	CUDA_CHECK(hipFree(dSum));
}