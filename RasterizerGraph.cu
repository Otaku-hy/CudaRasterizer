#include "hip/hip_runtime.h"
#include "GLFW/glfw3.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "Rasterizer.cuh"
#include "RasterizerGraph.h"
#include "RasterUnitFunction.cuh"
#include "RasterParallelAlgorithm.cuh"


namespace
{
	hipStream_t workStreams[16];
	hipStream_t renderingStream;
	hipStream_t glStream;

	hipGraph_t renderGraph = nullptr;
	hipGraphExec_t graphInstance = nullptr;

	hipEvent_t events[16];
}

using namespace CRPipeline;

void InitializeCudaGraph()
{
	for (int i = 0; i < 16; i++)
	{
		CUDA_CHECK(hipStreamCreateWithFlags(&workStreams[i], hipStreamNonBlocking));
		CUDA_CHECK(hipEventCreate(&events[i]));
	}
	CUDA_CHECK(hipStreamCreateWithFlags(&renderingStream, hipStreamNonBlocking));
	CUDA_CHECK(hipStreamCreateWithFlags(&glStream, hipStreamNonBlocking));
}

void CleanupCudaGraph()
{
	for (int i = 0; i < 16; i++)
	{
		CUDA_CHECK(hipStreamDestroy(workStreams[i]));
		CUDA_CHECK(hipEventDestroy(events[i]));
	}
	CUDA_CHECK(hipStreamDestroy(renderingStream));
	CUDA_CHECK(hipStreamDestroy(glStream));

	if (renderGraph != nullptr)
	{
		CUDA_CHECK(hipGraphDestroy(renderGraph));
		renderGraph = nullptr;
	}
	if (graphInstance != nullptr)
	{
		CUDA_CHECK(hipGraphExecDestroy(graphInstance));
		graphInstance = nullptr;
	}
}

void BeginFrame()
{

	// Reset allocators
	hipStreamWaitEvent(workStreams[2], events[0], 0); // Wait for SetGraphicsRoot
	CUDA_CHECK(hipMemsetAsync(dChunkAllocator, 0, sizeof(unsigned int), workStreams[2]));
	CUDA_CHECK(hipEventRecord(events[2], workStreams[2]));

	hipStreamWaitEvent(workStreams[3], events[0], 0); // Wait for SetGraphicsRoot
	CUDA_CHECK(hipMemsetAsync(dTileChunkAllocator, 0, sizeof(unsigned int), workStreams[3]));
	CUDA_CHECK(hipEventRecord(events[3], workStreams[3]));

	hipStreamWaitEvent(workStreams[4], events[0], 0); // Wait for SetGraphicsRoot
	CUDA_CHECK(hipMemsetAsync(dQuadAllocator, 0, sizeof(unsigned int), workStreams[4]));
	CUDA_CHECK(hipEventRecord(events[4], workStreams[4]));

	hipStreamWaitEvent(workStreams[5], events[0], 0); // Wait for SetGraphicsRoot
	CUDA_CHECK(hipMemcpyAsync(dSubTriangleCounter, hdcPrimitiveCount, sizeof(unsigned int), hipMemcpyHostToDevice, workStreams[5]));
	CUDA_CHECK(hipEventRecord(events[5], workStreams[5]));

	// Clear depth buffer & render target & hiz
	hipStreamWaitEvent(workStreams[6], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dFragmentStream, 0, sizeof(FragmentPSin) * windowWidth * windowHeight * AVERAGE_OVERDRAW, workStreams[6]);
	hipEventRecord(events[6], workStreams[6]);

	hipStreamWaitEvent(workStreams[7], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dRenderTarget, 0, sizeof(float4) * windowHeight * windowWidth, workStreams[7]);
	hipEventRecord(events[7], workStreams[7]);

	hipStreamWaitEvent(workStreams[8], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dHiZ, 0xFF, sizeof(unsigned) * MAX_BIN_COUNT * MAX_TILE_COUNT, workStreams[8]);
	hipEventRecord(events[8], workStreams[8]);

	hipStreamWaitEvent(workStreams[9], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dDepthBuffer, 0xFF, sizeof(unsigned int) * windowWidth * windowHeight, workStreams[9]);
	hipEventRecord(events[9], workStreams[9]);

	// Clear inner buffers
	hipStreamWaitEvent(workStreams[10], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dPrimitiveStream, 0xFF, sizeof(Primitive) * dcPrimitiveCount * 4, workStreams[10]);
	hipEventRecord(events[10], workStreams[10]);

	hipStreamWaitEvent(workStreams[12], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dPixelBaseIdx, 0xFF, sizeof(int) * windowWidth * windowHeight, workStreams[12]);
	hipEventRecord(events[12], workStreams[12]);

	// Clear queue counters
	hipStreamWaitEvent(workStreams[13], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dSubQueuePrimCount, 0, sizeof(unsigned) * MAX_BINNING_WAVE * BINNING_STAGE_BLOCK_SIZE, workStreams[13]);
	hipEventRecord(events[13], workStreams[13]);

	hipStreamWaitEvent(workStreams[14], events[0], 0); // Wait for SetGraphicsRoot
	hipMemsetAsync(dTileQueuePrimCount, 0, sizeof(unsigned) * MAX_BIN_COUNT * MAX_TILE_COUNT * TILE_QUEUE_ENTRY, workStreams[14]);
	hipEventRecord(events[14], workStreams[14]);
}

void EndFrame()
{
}

void RenderPipeline(Texture2D tex)
{
	const int threadsPerBlock = 256;

	// ===== VERTEX SHADING STAGE =====
	{
		int blocksPerGrid = (dcVertexCount + threadsPerBlock - 1) / threadsPerBlock;
		VertexFetchAndShading << <blocksPerGrid, threadsPerBlock, 0, renderingStream >> > (dcVertexCount, dInVertexStream, dOutVertexStream);
	}

	// ===== PRIMITIVE ASSEMBLY STAGE =====
	{
		CUDA_CHECK(hipStreamWaitEvent(renderingStream, events[5], 0)); // Wait for SubTriangleCounter memcpy
		hipStreamWaitEvent(renderingStream, events[10], 0); // Wait for PrimitiveStream memset
		int blocksPerGrid = (dcPrimitiveCount + threadsPerBlock - 1) / threadsPerBlock;
		PrimitiveAssembly << <blocksPerGrid, threadsPerBlock, 0, renderingStream >> > (dcPrimitiveCount, dIndexStream, dOutVertexStream,
			dPrimitiveStream, windowWidth, windowHeight, dSubTriangleCounter);
	}

	// ===== PRIMITIVE COMPACTION STAGE =====
	{
		PrimitiveCompaction(AVERAGE_PRIMITIVE_CULLED_COUNT * dcPrimitiveCount, dPrimitiveStream, dCompactedPrimitiveStream, dPrimitiveCounter, renderingStream);
		hipEventRecord(events[1], renderingStream);
	}

	// ===== TRIANGLE SETUP STAGE &  BINNING STAGE=====
	{
		unsigned compactedCount = std::max(1u * dcPrimitiveCount * AVERAGE_PRIMITIVE_CULLED_COUNT >> 1, 2048u);
		int blocksPerGrid = (compactedCount + threadsPerBlock - 1) / threadsPerBlock;
		hipStreamWaitEvent(workStreams[15], events[1], 0); // Wait for PrimitiveCompaction
		TriangleSetup << <blocksPerGrid, threadsPerBlock, 0, workStreams[15] >> > (dPrimitiveCounter, dCompactedPrimitiveStream, dTriSetupData);
		hipEventRecord(events[15], workStreams[15]);

		hipStreamWaitEvent(renderingStream, events[13], 0); // Wait for SubQueuePrimCount memset
		hipStreamWaitEvent(renderingStream, events[2], 0); // Wait for ChunkAllocator memset
		PrimitiveBinning << <blocksPerGrid, threadsPerBlock, 0, renderingStream >> > (dPrimitiveCounter, dCompactedPrimitiveStream, dChunkAllocator,
			dSubQueueBaseIndex, dSubQueuePrimCount, dBinQueue, windowWidth, windowHeight);
	}

	// ===== COARSE RASTERIZATION STAGE =====
	{
		int xUpper = UPPER_BOUND(windowWidth, BIN_PIXEL_SIZE_LOG2);
		int yUpper = UPPER_BOUND(windowHeight, BIN_PIXEL_SIZE_LOG2);
		dim3 blockSize(xUpper >> BIN_PIXEL_SIZE_LOG2, yUpper >> BIN_PIXEL_SIZE_LOG2);
		hipStreamWaitEvent(renderingStream, events[14], 0); // Wait for TileQueuePrimCount memset
		hipStreamWaitEvent(renderingStream, events[3], 0); // Wait for TileChunkAllocator memset
		hipStreamWaitEvent(renderingStream, events[8], 0); // Wait for HiZ memset
		hipStreamWaitEvent(renderingStream, events[15], 0); // Wait for TriangleSetup
		CoarseRasterizer << <blockSize, dim3(16, 16), 0, renderingStream >> > (dPrimitiveCounter, dSubQueueBaseIndex, dSubQueuePrimCount, dBinQueue,
			dTriSetupData, dHiZ, dTileChunkAllocator, dTileQueueBaseIndex, dTileQueuePrimCount, dTileQueue, windowWidth, windowHeight);
	}

	// ===== FINE RASTERIZATION STAGE =====
	{
		int tileXUpper = UPPER_BOUND(windowWidth, TILE_PIXEL_SIZE_LOG2) >> TILE_PIXEL_SIZE_LOG2;
		int tileYUpper = UPPER_BOUND(windowHeight, TILE_PIXEL_SIZE_LOG2) >> TILE_PIXEL_SIZE_LOG2 >> 2;
		hipStreamWaitEvent(renderingStream, events[4], 0); // Wait for QuadAllocator memset
		hipStreamWaitEvent(renderingStream, events[9], 0); // Wait for DepthBuffer memset
		hipStreamWaitEvent(renderingStream, events[6], 0); // Wait for FragmentStream memset
		FineRasterizerWIP << <dim3(tileXUpper, tileYUpper), dim3(32, FINE_RASTER_TILE_PER_BLOCK), 0, renderingStream >> > (dTileQueueBaseIndex, dTileQueuePrimCount, dTileQueue,
			dTriSetupData, dDepthBuffer, dQuadAllocator, dFragmentStream, windowWidth, windowHeight);
	}

	// ===== PIXEL SHADING STAGE =====
	{
		int blocksPerGrid = (windowWidth * windowHeight * AVERAGE_OVERDRAW + threadsPerBlock - 1) / threadsPerBlock;
		hipStreamWaitEvent(renderingStream, events[12], 0); // wait for PixelBaseIdx memset
		PixelShader << <blocksPerGrid, threadsPerBlock, 0, renderingStream >> > (dQuadAllocator, dFragmentStream, dFragmentOutStream, dPixelBaseIdx, windowWidth, windowHeight, tex);
	}

	// ===== ROP STAGE =====
	{
		dim3 blockSize(16, 16);
		dim3 gridSize((windowWidth + blockSize.x - 1) / blockSize.x, (windowHeight + blockSize.y - 1) / blockSize.y);
		hipStreamWaitEvent(renderingStream, events[7], 0); // Wait for RenderTarget memset
		ROP << <gridSize, blockSize, 0, renderingStream >> > (dFragmentOutStream, dPixelBaseIdx, windowWidth, windowHeight, dDepthBuffer, dRenderTarget);
	}


}

void BuildPipeline(unsigned char* cudaMappedRT, unsigned* depthBuffer,
	const VertexVSIn* vertexStream, const uint32_t* indexStream,
	int indexCount, int vertexCount, MatricesCBuffer* cb, Texture2D tex)
{
	CUDA_CHECK(hipStreamBeginCapture(renderingStream, hipStreamCaptureModeGlobal));

	hipEventRecord(events[0], renderingStream);
	SetGraphicsRoot(cudaMappedRT, depthBuffer,
		vertexStream, indexStream,
		indexCount, vertexCount, cb, renderingStream);
	hipEventRecord(events[1], renderingStream);
	BeginFrame();
	RenderPipeline(tex);
	EndFrame();

	CUDA_CHECK(hipStreamEndCapture(renderingStream, &renderGraph));
	CUDA_CHECK(hipGraphInstantiate(&graphInstance, renderGraph, nullptr, nullptr, 0));
}

void RasterizeWithGraph(GLuint rtBuffer, unsigned* depthBuffer,
	const VertexVSIn* vertexStream, const uint32_t* indexStream,
	int indexCount, int vertexCount, MatricesCBuffer* cb, Texture2D tex)
{
	hipEvent_t renderPipeEvent;
	CUDA_CHECK(hipEventCreate(&renderPipeEvent));

	CUDA_CHECK(hipGraphLaunch(graphInstance, renderingStream));
	CUDA_CHECK(hipEventRecord(renderPipeEvent, renderingStream));

	unsigned char* cudaMappedRT = nullptr;
	CUDA_CHECK(cudaGLMapBufferObjectAsync((void**)&cudaMappedRT, rtBuffer, glStream));

	int blocksPerGrid = (windowWidth * windowHeight + 255) / 256;
	hipStreamWaitEvent(glStream, renderPipeEvent, 0); // Wait for RT mapping
	StreamingToFrameBuffer << <blocksPerGrid, 256, 0, glStream >> > (windowWidth * windowHeight, dRenderTarget, cudaMappedRT, windowWidth);

	CUDA_CHECK(cudaGLUnmapBufferObjectAsync(rtBuffer, glStream));

	CUDA_CHECK(hipStreamSynchronize(glStream));
}
